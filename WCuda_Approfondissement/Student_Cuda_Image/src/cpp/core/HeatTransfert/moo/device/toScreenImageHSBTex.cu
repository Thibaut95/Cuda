#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "DomaineMath_GPU.h"
#include "IndiceTools_GPU.h"
#include "Calibreur_GPU.h"
#include "ColorTools_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public		 	*|
 \*-------------------------------------*/
 texture<float,2,hipReadModeElementType> textureAB;
__global__ void toScreenImageHSBTex(uchar4* ptrDevPixels, uint w, uint h, Calibreur<float> calibreur);
__host__ void initTextureABHSB(float* ptrDevImageAB, int w, int h);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

 __host__ void initTextureAB(float* ptrDevImageAB, int w, int h)
 {
    textureAB.addressMode[0]=hipAddressModeClamp;
    textureAB.addressMode[1]=hipAddressModeClamp;
    textureAB.filterMode=hipFilterModePoint;
    textureAB.normalized=false;

 size_t pitch = w * sizeof(float);
 hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
 HANDLE_ERROR(hipBindTexture2D(NULL,textureAB,ptrDevImageAB,channelDesc,w,h,pitch));
 }

__global__ void toScreenImageHSBTex(uchar4* ptrDevPixels, uint w, uint h, Calibreur<float> calibreur)
    {
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int i;
    int j;

    int s = TID;

    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);

	float hue=tex2D(textureAB,j,i);
	calibreur.calibrer(&hue);
	ColorTools::HSB_TO_RVB(hue, &ptrDevPixels[s]);

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

