#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "DomaineMath_GPU.h"
#include "IndiceTools_GPU.h"
#include "Calibreur_GPU.h"
#include "ColorTools_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public		 	*|
 \*-------------------------------------*/
 texture<float,2,hipReadModeElementType> textureAB;
__global__ void diffusionTex(float* ptrDevOutput, uint w, uint h);
__host__ void initTextureAB(float* ptrDevImageAB, int w, int h);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
 __host__ void initTextureAB(float* ptrDevImageAB, int w, int h)
     {
     textureAB.addressMode[0]=hipAddressModeClamp;
     textureAB.addressMode[1]=hipAddressModeClamp;
     textureAB.filterMode=hipFilterModePoint;
     textureAB.normalized=false;

     size_t pitch = w * sizeof(float);
     hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
     HANDLE_ERROR(hipBindTexture2D(NULL,textureAB,ptrDevImageAB,channelDesc,w,h,pitch));
     }

__global__ void diffusionTex(float* ptrDevOutput, uint w, uint h)
    {
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = (w-2) * (h-2);

    int i;
    int j;

    int s = TID;

    while (s < WH)
	{
	int sReal = s+w+1;
	IndiceTools::toIJ(sReal, w, &i, &j);

	ptrDevOutput[sReal]=tex2D(textureAB,j,i)+0.25*(tex2D(textureAB,j+1,i)+tex2D(textureAB,j,i+1)+tex2D(textureAB,j-1,i)+tex2D(textureAB,j,i-1)-4*tex2D(textureAB,j,i));

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

