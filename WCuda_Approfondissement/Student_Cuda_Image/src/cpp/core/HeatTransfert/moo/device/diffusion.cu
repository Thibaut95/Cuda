#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "DomaineMath_GPU.h"
#include "IndiceTools_GPU.h"
#include "Calibreur_GPU.h"
#include "ColorTools_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public		 	*|
 \*-------------------------------------*/

__global__ void diffusion(float* ptrDevInput, float* ptrDevOutput, uint w, uint h);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void diffusion(float* ptrDevInput, float* ptrDevOutput, uint w, uint h)
    {
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = (w-2) * (h-2);

    int s = TID;

    while (s < WH)
	{
	int sReal = s+w+1;
	ptrDevOutput[sReal]=ptrDevInput[sReal]+0.25*(ptrDevInput[sReal+1]+ptrDevInput[sReal-1]+ptrDevInput[sReal+w]+ptrDevInput[sReal-w]-4*ptrDevInput[sReal]);

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

