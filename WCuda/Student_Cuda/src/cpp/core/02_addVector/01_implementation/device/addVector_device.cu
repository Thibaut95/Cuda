#include "hip/hip_runtime.h"
#include <Indice2D.h>
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n);
__global__ void addVector11(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 * pattern entrelacement
 */
__global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n)
    {

    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    // Debug, facultatif
    //    if (TID == 0)
    //	{
    //	printf("Coucou from device tid = %d", TID); //required   Device::synchronize(); after the call of kernel
    //	}

    //TODO addVector pattern entrelacement
    int s = TID;
    while (s < n)
	{
	ptrDevW[s] = ptrDevV1[s] + ptrDevV2[s];
	s += NB_THREAD;
	}
    }

/**
 * pattern 1<-->1
 * hyp: #thread=#caseVecteur=n
 */
__global__ void addVector11(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n)
    {
    // TODO addVector pattern  1<-->1
    const int TID = Indice2D::tid();
    ptrDevW[TID] = ptrDevV1[TID] + ptrDevV2[TID];
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

