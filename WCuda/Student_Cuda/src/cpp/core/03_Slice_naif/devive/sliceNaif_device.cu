#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void reductionIntraThread(float* tabGM, int nbSlice);
__global__ void ecrasementGM(float* tabGM, int moitier);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ float aireRectangle(float xs, int nbSlice);
static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * Chaque thread effecteur une reduction avec le patern d'entrelacement,
 * puis stocke son résultat dans SA case dans tabGM
 * tabGM est un tableau promu, qui a autant de case que de thread
 */
__global__ void reductionIntraThread(float* tabGM, int nbSlice)
    {
    // TODO SliceNaif
    const int TID = Indice2D::tid();
    int s = TID;
    const int NB_THREAD = Indice2D::nbThread();
    float sum = 0.0;
    float xi;
    while (s < nbSlice)
	{
	xi = s / (float) nbSlice;
	sum += (1 / (float) nbSlice) * f(xi);
	s += NB_THREAD;
	}
    tabGM[TID] = sum;
    }

/**
 * Pour version NaivePLus
 *
 * Ramene la semantique du tableau pour une reduction, dans la premiere moitier, par ecraseemt
 * Doit etre appeler plusieurs fois coter host, pour tout ramener dans tabGM[0]
 */
__global__ void ecrasementGM(float* tabGM, int moitier)
    {
    // TODO SliceNaif
    const int TID = Indice2D::tid();
    int s = TID;
    const int NB_THREAD = Indice2D::nbThread();
    while (s < moitier)
	{
	tabGM[s] += tabGM[moitier + s];
	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*-----------------------*\
 | Reduction Intrathread  *|
 \*-----------------------*/

__device__ float aireRectangle(float xs, int nbSlice)
    {
    // TODO SliceNaif
    return ((float) 1 / (float) nbSlice) * f(xs);
    }

__device__ float f(float x)
    {
    // TODO SliceNaif
    return 4 / (float) (1 + x * x);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

