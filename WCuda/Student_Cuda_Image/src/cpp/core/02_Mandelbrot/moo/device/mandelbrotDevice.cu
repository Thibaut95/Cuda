#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "MandelbrotMath.h"  // a decommenter une fois coder

#include "DomaineMath_GPU.h"
#include "IndiceTools_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public		 	*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, int n)
    {
    // TODO Mandelbrot GPU :
    // build MandelbrotMath
    // entrelacement
    MandelbrotMath mandelbrotMath = MandelbrotMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int i;
    int j;

    int s = TID;
    // TODO Rippling GPU  pattern entrelacement
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);

	mandelbrotMath.colorIJ(&ptrDevPixels[s], i, j, n, domaineMath);

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

