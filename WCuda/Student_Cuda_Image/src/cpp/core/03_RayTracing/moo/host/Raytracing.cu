#include "hip/hip_runtime.h"
#include <iostream>

#include "Raytracing.h"
#include "Device.h"
#include "SphereCreator.h"
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/


/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres);
__global__ void kernelRaytacingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres);
__global__ void kernelRaytacingCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSpheres);

__host__ void uploadToCM(Sphere* ptrTabSpheres, int nbSpheres);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

Raytracing::Raytracing(const Grid& grid, uint w, uint h, float dt, int nbSpheres,const MemoryType& memoryType) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda_RGBA_uchar4"), memoryType(memoryType)
    {
    // Inputs
    this->dt = dt;
    this->nbSpheres = nbSpheres;

    // Tools
    this->sizeSpheres = nbSpheres * sizeof(Sphere);
    this->t = 0;
    Device::malloc(&ptrDevTabSpheres, sizeSpheres);

    // spheres
	{
	SphereCreator sphereCreator(nbSpheres, w, h);
	uploadToDevice(sphereCreator.getTabSphere());
	} // SphereCreator depiler, donc detruit, tabSphere cote host detruit!

    cout << "[Raytracing] : memoryType : " << memoryType;
    }

Raytracing::~Raytracing()
    {
    // TODO Raytracing GPU MemoryManagement free
    switch (memoryType)
	{
	case GM:
	    {
	    Device::free(ptrDevTabSpheres);
	    // ??
	    break;
	    }
	case CM:
	    {
	    assert(false);// to delete once implement
	    // ??
	    break;
	    }
	case SM:
	    {
	    Device::free(ptrDevTabSpheres);
	    // ??
	    break;
	    }
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * Override
 */
void Raytracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    // TODO Raytracing GPU lauch kernel

    switch (memoryType)
	{
	case GM:
	    {
	    kernelRaytacingGM<<<dg,db>>>(ptrDevPixels,w,h,t,ptrDevTabSpheres,nbSpheres);
	    // Call the kernel kernelRaytacingGM (line 25)
	    break;
	    }
	case CM:
	    {
	    kernelRaytacingCM<<<dg,db>>>(ptrDevPixels,w,h,t,nbSpheres);
	    // Call the kernel kernelRaytacingCM (line 26)
	    break;
	    }
	case SM:
	    {

	    kernelRaytacingSM<<<dg,db,sizeSpheres>>>(ptrDevPixels,w,h,t,ptrDevTabSpheres,nbSpheres);
	    // Call the kernel kernelRaytacingSM (line 27)
	    break;
	    }
	}
    }

/**
 * Override
 */
void Raytracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

void Raytracing::uploadToDevice(Sphere* ptrTabSpheres)
    {
    // TODO Raytracing GPU MemoryManagement
    switch (memoryType)
	{
	case GM:
	    {
	    Device::memcpyHToD(ptrDevTabSpheres, ptrTabSpheres, sizeSpheres);
	    break;
	    }
	case CM:
	    {
	    uploadToCM(ptrTabSpheres, nbSpheres);
	    break;
	    }
	case SM:
	    {
	    Device::memcpyHToD(ptrDevTabSpheres, ptrTabSpheres, sizeSpheres);
	    break;
	    }
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

