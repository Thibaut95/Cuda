#include "hip/hip_runtime.h"
#include <assert.h>

#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "RaytracingMath.h"
#include "Sphere.h"
#include "nbSphere.h"

#include "IndiceTools_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

__constant__ Sphere TAB_SPHERES_CM[NB_SPHERE]; // NB_SPHERE define in nbSphere.h

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres);
__global__ void kernelRaytacingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres);
__global__ void kernelRaytacingCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSpheres);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSpheres, int nbSpheres);
static __device__ void copyGMtoSM(Sphere* tabGM, Sphere* tabSM, int nbSpheres);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		host			*|
 \*-------------------------------------*/

/**
 * must be called by host
 * ptrTabSpheres est un tableau de sphere cote host
 */
__host__ void uploadToCM(Sphere* ptrTabSpheres, int nbSpheres)
    {
    assert(nbSpheres == NB_SPHERE);

    // TODO Raytracing GPU CM
    // mettre ptrTabSpheres dans TAB_SPHERES_CM (line 16)
    size_t size = NB_SPHERE * sizeof(Sphere);
    int offset = 0;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TAB_SPHERES_CM), ptrTabSpheres, size, offset, hipMemcpyHostToDevice));
    }

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres)
    {
    // TODO Raytracing GPU GM
    // call work with good input
    work(ptrDevPixels, w, h, t, ptrTabSpheresGM, nbSpheres);
    }

__global__ void kernelRaytacingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrTabSpheresGM, int nbSpheres)
    {
    // TODO Raytracing GPU SM
    // call work with good input
    extern  __shared__  Sphere tabSM[];
    copyGMtoSM(ptrTabSpheresGM, tabSM, nbSpheres);
    __syncthreads();
    work(ptrDevPixels, w, h, t, tabSM, nbSpheres);
    }

__global__ void kernelRaytacingCM(uchar4* ptrDevPixels, uint w, uint h, float t, int nbSpheres)
    {
    // TODO Raytracing GPU CM
    // call work with good input
    work(ptrDevPixels, w, h, t, TAB_SPHERES_CM, nbSpheres);
    }

/*--------------------------------------*\
 |*		private			*|
 \*-------------------------------------*/

/**
 * Methode commune au 3 kernel ci-dessus.
 * Ici on ne sait pas si derriere ptrDevTabSpheres, c'est
 * 	- de la GM?
 * 	- de la SM?
 * 	- de la CM?
 * Pas d'importance, c'est un pointeur et on travail avec!
 */
__device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSpheres, int nbSpheres)
    {
    // TODO Raytracing GPU device side
    // create RaytracingMath
    // entrelacement
    RaytracingMath rayTracingMath = RaytracingMath(ptrDevTabSpheres, nbSpheres);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int i;
    int j;

    int s = TID;
    // TODO Rippling GPU  pattern entrelacement
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);

	rayTracingMath.color(&ptrDevPixels[s], i, j, t);

	s += NB_THREAD;
	}
    }

__device__ void copyGMtoSM(Sphere* tabGM, Sphere* tabSM, int nbSpheres)
    {
	const int TID = Indice2D::tidLocal();
        const int NB_THREAD_LOCAL = Indice2D::nbThreadLocal();

        int s = TID;

        while(s<nbSpheres)
            {
            tabSM[s]=tabGM[s];
            s += NB_THREAD_LOCAL;
            }
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

