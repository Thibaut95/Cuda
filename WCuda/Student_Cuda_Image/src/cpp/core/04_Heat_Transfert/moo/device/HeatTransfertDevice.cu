#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "HeatTransfertMath.h"  // a decommenter une fois coder

#include "DomaineMath_GPU.h"
#include "IndiceTools_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public		 	*|
 \*-------------------------------------*/

__global__ void heatTransfert(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void heatTransfert(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, int n)
    {
    // TODO Mandelbrot GPU :
    // build MandelbrotMath
    // entrelacement
    HeatTransfertMath heatTransfertMath = HeatTransfertMath(n);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int i;
    int j;

    int s = TID;
    // TODO Rippling GPU  pattern entrelacement
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);

	heatTransfertMath.colorIJ(&ptrDevPixels[s], i, j, n, domaineMath);

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

