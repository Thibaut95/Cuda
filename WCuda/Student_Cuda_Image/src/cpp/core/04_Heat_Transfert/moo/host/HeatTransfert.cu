#include "hip/hip_runtime.h"
#include "HeatTransfert.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void heatTransfert(uchar4* ptrDevPixels, uint w, uint h,DomaineMath domaineMath, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

HeatTransfert::HeatTransfert(const Grid& grid, uint w, uint h, const DomaineMath& domaineMath, int nMin, int nMax) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_Cuda_RGBA_uchar4", domaineMath), variateurT(Interval<int>(nMin, nMax), 1)
    {
    // Inputs/animation
    this->nMin = nMin;
    this->nMax = nMax;

    // Tools
    this->t = nMin;
    }

HeatTransfert::~HeatTransfert()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void HeatTransfert::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    int n = t;

     // TODO Mandelbrot GPU
    // lauch kernel (line 18)
    heatTransfert<<<dg,db>>>(ptrDevPixels, w, h, domaineMath, n);
    }

/**
 * Override
 * Call periodicly by the API
 */
void HeatTransfert::animationStep()
    {
    this->t = variateurT.varierAndGet();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

