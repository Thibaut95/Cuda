#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "RipplingMath.h"

//#include "RipplingMath.h" // a decommenter une fois coder

#include "IndiceTools_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, uint w, uint h, float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rippling(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    // TODO instacier RipplingMath
    RipplingMath ripplingMath = RipplingMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int i;
    int j;

    int s = TID;
    // TODO Rippling GPU  pattern entrelacement
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);

	ripplingMath.colorIJ(&ptrDevPixels[s], i, j, t);

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

